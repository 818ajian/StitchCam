#include "hip/hip_runtime.h"
#include <>
#include <opencv2/gpu/device/saturate_cast.hpp>

#include "MyBlender.h"

using namespace std;

template<typename T>
__global__ void kernelFeed(int rows, int cols, T *dst, const T *src, const uchar *mask, int dStep, int sStep, int mStep)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = x + y * mStep;
	if (x < cols && y < rows && mask[offset])
	{
		int dOffset = x * 3 + y * dStep;
		offset = x * 3 + y * sStep;
		dst[dOffset] = src[offset];
		dst[dOffset + 1] = src[offset + 1];
		dst[dOffset + 2] = src[offset + 2];
	}
}

namespace cv {
namespace detail {

	void cudaFeed(const gpu::GpuMat &image, const gpu::GpuMat &mask, gpu::GpuMat &dst, int dx, int dy)
	{
		dim3 threads(16, 16);	// 256 threads yealds better performance
		dim3 blocks(image.cols / threads.x, image.rows / threads.y);
		switch (image.type())
		{
		case CV_8UC3:
			kernelFeed<<<blocks, threads>>>(image.rows, image.cols, 
				dst.ptr<uchar>(dy) + dx * 3, image.ptr<uchar>(), mask.ptr<uchar>(),
				dst.step, image.step, mask.step);
			hipDeviceSynchronize();
			break;
		case CV_16SC3:
			kernelFeed<<<blocks, threads>>>(image.rows, image.cols, 
				dst.ptr<short>(dy) + dx * 3, image.ptr<short>(), mask.ptr<uchar>(),
				dst.step, image.step, mask.step);
			hipDeviceSynchronize();
			break;
		}
	}

	void cudaFeed(const Mat &image, const Mat &mask, gpu::GpuMat &dst, int dx, int dy)
	{
		gpu::GpuMat gpuImg;
		gpu::GpuMat gpuMask;
		gpuImg.upload(image);
		gpuMask.upload(mask);
		cudaFeed(gpuImg, gpuMask, dst, dx, dy);
	}

}	// namespace detail
}	// namespace cv;
