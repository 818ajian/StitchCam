#include "hip/hip_runtime.h"
#include <opencv2/gpu/device/saturate_cast.hpp>

#include "MyCompensator.h"


using namespace std;

namespace cv {
namespace gpu {
namespace device {

//////////////////////////////////////////////////////////////////////////
// Exposure compensate kernel
template<typename T>
__global__ void applyKernel(int rows, int cols, T *ptr, int step, double scale)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = x * 3 + y * step;
	if (x < cols && y < rows)
    {
		ptr[offset] = saturate_cast<T>(scale * ptr[offset]);
		ptr[offset + 1] = saturate_cast<T>(scale * ptr[offset + 1]);
		ptr[offset + 2] = saturate_cast<T>(scale * ptr[offset + 2]);
    }
}


void cudaApply(gpu::GpuMat &image, double scale)
{
	int rows = image.rows;
	int cols = image.cols;
#ifdef	JETSON_TK1
	dim3 threads(8, 8);	// 64 threads for Jetson TK1
#else
	dim3 threads(16, 16);	// 256 threads yealds better performance
#endif
	dim3 blocks(cols / threads.x, rows / threads.y);

	switch (image.type())
	{
	case CV_8UC3:
		applyKernel<<<blocks, threads>>>(rows, cols, (uchar *)image.datastart, image.step, scale);
		hipDeviceSynchronize();
		break;
	case CV_16SC3:
		applyKernel<<<blocks, threads>>>(rows, cols, (short *)image.datastart, image.step, scale);
		hipDeviceSynchronize();
		break;
	}
}

void cudaApply(Mat &image, double scale)
{
	gpu::GpuMat gpuMat;
	gpuMat.upload(image);
	cudaApply(gpuMat, scale);
	gpuMat.download(image);
}
}	// namespace device
}	// namespace gpu
}	// namespace cv;
