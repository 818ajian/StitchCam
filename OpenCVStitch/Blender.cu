#include "hip/hip_runtime.h"
#include ""
#include <opencv2/gpu/device/saturate_cast.hpp>

#include "../MyBlender.h"

#include <stdio.h>

using namespace std;

__global__ void kernelFeed(int rows, int cols, uchar *dst, const uchar *src, const uchar *mask, int dStep, int sStep, int mStep)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = x + y * mStep;
	if (x < cols && y < rows && mask[offset])
	{
		int dOffset = x * 3 + y * dStep;
		offset = x * 3 + y * sStep;
		dst[dOffset] = src[offset];
		dst[dOffset + 1] = src[offset + 1];
		dst[dOffset + 2] = src[offset + 2];
	}
}

namespace cv {
namespace detail {

	void MyBlender::cudaFeed(const gpu::GpuMat &image, const gpu::GpuMat &mask, int dx, int dy)
	{
		dim3 threads(16, 16);	// 256 threads yealds better performance
		dim3 blocks(image.cols / threads.x, image.rows / threads.y);
		kernelFeed<<<blocks, threads>>>(image.rows, image.cols, 
			gpuDst_.ptr<uchar>(dy) + dx * 3, image.ptr<uchar>(), mask.ptr<uchar>(), 
			gpuDst_.step, image.step, mask.step);
	}

	void MyBlender::cudaFeed(const Mat &image, const Mat &mask, int dx, int dy)
	{
		gpu::GpuMat gpuImg;
		gpu::GpuMat gpuMask;
		gpuImg.upload(image);
		gpuMask.upload(mask);
		cudaFeed(gpuImg, gpuMask, dx, dy);
	}

}	// namespace detail
}	// namespace cv;