#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <opencv2/gpu/device/saturate_cast.hpp>

#include "MyCompensator.h"

//#include <stdio.h>

using namespace std;

namespace cv {
namespace gpu {
namespace device {

//////////////////////////////////////////////////////////////////////////
// Exposure compensate kernel
template<typename T>
__global__ void applyKernel(int rows, int cols, T *ptr, int step, float scale)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = x * 3 + y * step;
	if (x < cols && y < rows)
    {
		ptr[offset] = saturate_cast<T>(scale * ptr[offset]);
		ptr[offset + 1] = saturate_cast<T>(scale * ptr[offset + 1]);
		ptr[offset + 2] = saturate_cast<T>(scale * ptr[offset + 2]);
    }
}


void cudaApply(gpu::GpuMat &image, double scale)
{
	int rows = image.rows;
	int cols = image.cols;
	dim3 threads(16, 16);	// 256 threads yealds better performance
	dim3 grid(cols / threads.x, rows / threads.y);

	switch (image.type())
	{
	case CV_8UC3:
		applyKernel<<<grid, threads>>>(rows, cols, (uchar *)image.datastart, image.step, scale);
		hipDeviceSynchronize();
		break;
	case CV_16SC3:
		applyKernel<<<grid, threads>>>(rows, cols, (short *)image.datastart, image.step, scale);
		hipDeviceSynchronize();
		break;
	}
}

void cudaApply(Mat &image, double scale)
{
	gpu::GpuMat gpuMat;
	gpuMat.upload(image);
	cudaApply(gpuMat, scale);
	gpuMat.download(image);
}
}	// namespace device
}	// namespace gpu
}	// namespace cv;
